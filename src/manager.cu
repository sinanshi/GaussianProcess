/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <manager.hh>

gpuPrepareLikelihood::gpuPrepareLikelihood(float *Q_, float *targets_, int N) {
      cudaStatus = hipGetDevice(0);
      cusolverStatus = hipsolverDnCreate(&handle);
      hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

      Q = (float *)malloc(sizeof(float) * N * N);
      targets = (float *)malloc(sizeof(float) * N);

      Q = Q_; // point to Q
      targets = targets_;

      cudaStatus = hipMalloc((void **) &d_Q, N * N * sizeof(float));
      cudaStatus = hipMalloc((void **) &d_targets, N * sizeof(float));
//      cudaStatus = hipMalloc((void **) &d_invQ, N * sizeof(float));
//      cudaStatus = hipMalloc((void **) &d_info, sizeof(int));
      cudaStatus = hipMemcpy(d_Q, Q, N * N * sizeof(float), hipMemcpyHostToDevice); 
      cudaStatus = hipMemcpy(d_targets, targets, N * sizeof(float), hipMemcpyHostToDevice); 
}


gpuPrepareLikelihood::~gpuPrepareLikelihood(){

//  cudaStatus = hipDeviceSynchronize(); // should be used for the timing.
  cudaStatus = hipFree(dev_L);
  cudaStatus = hipFree(d_invQ);
  cudaStatus = hipFree(d_invQt);
  cusolverStatus = hipsolverDnDestroy(handle);
  cudaStatus = hipDeviceReset();
  
  cudaStatus = hipFree(Work);
  free(invQ);
  free(invQt);
  free(logdetQ);
}


void gpuPrepareLikelihood::gpu_cholesky() {

  int *d_info, Lwork; //device version of info, worksp.size

//  int info_gpu = 0;

  hipMalloc((void **) &d_info, sizeof(int));

  // compute workspace size and prepare workspace
  cusolverStatus = hipsolverDnSpotrf_bufferSize(handle, uplo, N, d_Q, N, &Lwork);
  cudaStatus = hipMalloc((void**) &Work, Lwork * sizeof(float));

  cusolverStatus = hipsolverDnSpotrf(handle, uplo, N, d_Q, N, Work, Lwork, d_info);

  cusolverStatus = hipsolverDnSpotrs(handle, uplo, N, 1, d_Q, N, d_

  hipMemcpy(b, d_B, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);

}



//GPUAdder::GPUAdder (int* array_host_, int length_) {
//  array_host = array_host_;
//  length = length_;
//  int size = length * sizeof(int);
//  hipError_t err = hipMalloc((void**) &array_device, size);
//  assert(err == 0);
//  err = hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
//  assert(err == 0);
//}
//
//void GPUAdder::increment() {
//  kernel_add_one<<<64, 64>>>(array_device, length);
//  hipError_t err = hipGetLastError();
//  assert(err == 0);
//}
//
//void GPUAdder::retreive() {
//  int size = length * sizeof(int);
//  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
//  hipError_t err = hipGetLastError();
//  if(err != 0) { cout << err << endl; assert(0); }
//}
//
//void GPUAdder::retreive_to (int* array_host_, int length_) {
//  assert(length == length_);
//  int size = length * sizeof(int);
//  hipMemcpy(array_host_, array_device, size, hipMemcpyDeviceToHost);
//  hipError_t err = hipGetLastError();
//  assert(err == 0);
//}
//
//GPUAdder::~GPUAdder() {
//  hipFree(array_device);
//}
//
//
//
//
